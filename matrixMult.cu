#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>

// assumes square matrices. Each thread computes one element of P.
__global__ 
void matrixMultP(const float* M, const float* N, float* P, int width){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < width && col < width){
        float pValue = 0;
        int pIndex = row * width + col; // Flattened index of matrix P
        for (int k = 0; k < width; k++){
            int MIndex = row * width + k; // Flattened index of matrix M
            int NIndex = k * width + col; // Flattened index of matrix N
            pValue += M[MIndex] * N[NIndex];
        }
        P[pIndex] = pValue;
    }
    printf("Thread (%d, %d) computed element (%d, %d)\n", threadIdx.x, threadIdx.y, row, col);
}

//assumes square matrices. Each thread computes one row of P.
__global__
void matrixMultRow(const float* M, const float* N, float* P, int width){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < width){
        for (int col = 0; col < width; col++){
            float pValue = 0;
            int pIndex = row * width + col; // Flattened index of matrix P
            for (int k = 0; k < width; k++){
                int MIndex = row * width + k; // Flattened index of matrix M
                int NIndex = k * width + col; // Flattened index of matrix N
                pValue += M[MIndex] * N[NIndex];
            }
            P[pIndex] = pValue;
        }
    }
    printf("Thread (%d, %d) computed row %d\n", threadIdx.x, threadIdx.y, row);
}

int main(void){

   // initialize host matrices
    int width = 4;
    int size = width * width * sizeof(float);
    float *h_M = (float*)malloc(size);
    float *h_N = (float*)malloc(size);
    float *h_P = (float*)malloc(size);
    
    // populate matrices with random floats:
    for (int i = 0; i < width; i++){
        for (int j = 0; j < width; j++){
            h_M[i * width + j] = (float)rand() / RAND_MAX;
            h_N[i * width + j] = (float)rand() / RAND_MAX;
        }
    }


    // initialize device matrices
    float *d_M, *d_N, *d_P;
    hipMalloc((void**)&d_M, size);
    hipMalloc((void**)&d_N, size);
    hipMalloc((void**)&d_P, size);

    // copy host matrices to device
    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);

    // set dimension of grid (number of blocks)
    dim3 dimGrid(width/2, width/2, 1); // Splits matrix into 4 sections
    dim3 dimBlock(2, 2, 1); // each thread computes 1 element of P

    // launch kernel
    matrixMultP<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, width);

    // copy result back to host
    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);

    // verify result
    for (int i = 0; i < width; i++){
        for (int j = 0; j < width; j++){
            float pValue = 0;
            for (int k = 0; k < width; k++){
                pValue += h_M[i * width + k] * h_N[k * width + j];
            }
            if (fabs(pValue - h_P[i * width + j]) > 1e-5){
                fprintf(stderr, "Result verification failed at element (%d, %d)!\n", i, j);
                exit(EXIT_FAILURE);
            }
        }
    }

    printf("Test PASSED\n");

    // cleanup
    


    // each thread computes one row of P
    // set dimension of grid (number of blocks)
    dim3 dimGridR(2, 1, 1); // Splits matrix into 2 sections
    dim3 dimBlockR(2, 1, 1); // each thread computes 1 row of P

    // launch kernel
    matrixMultRow<<<dimGridR, dimBlockR>>>(d_M, d_N, d_P, width);

    // copy result back to host
    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);

    // verify result
    for (int i = 0; i < width; i++){
        for (int j = 0; j < width; j++){
            float pValue = 0;
            for (int k = 0; k < width; k++){
                pValue += h_M[i * width + k] * h_N[k * width + j];
            }
            if (fabs(pValue - h_P[i * width + j]) > 1e-5){
                fprintf(stderr, "Result verification failed at element (%d, %d)!\n", i, j);
                exit(EXIT_FAILURE);
            }
        }
    }

    printf("Test PASSED\n");


    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
    free(h_M);
    free(h_N);
    free(h_P);


}

