// Convolution where F is stored in F constant memory off chip. This slightly improves performance. OP/B = 0.5.

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>
#include <ctime>

// assign F to constant memory:
__constant__ float F[3 * 3]; // F is stored in constant memory off chip


// Kernel function. OP/B = 0.25. Horrible performance.
__global__ void convolution_2D_basic_kernel(const float *N, float *P, int r, int width, int height) {
    // Determine the row and column index of the output matrix.
    int outRow = blockIdx.y * blockDim.y + threadIdx.y;
    int outCol = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize the output value to 0.
    float Pvalue = 0.0f;
    for(int fRow = 0; fRow < 2 * r + 1; fRow++) {
        for(int fCol = 0; fCol < 2 * r + 1; fCol++) {
            int inRow = outRow - r + fRow;
            int inCol = outCol - r + fCol;
            if(inRow >= 0 && inRow < height && inCol >= 0 && inCol < width) {
                int N_index = inRow * width + inCol;
                int F_index = fRow * (2 * r + 1) + fCol;
                Pvalue += N[N_index] * F[F_index];
            }
        }
    }
    int P_index = outRow * width + outCol;
    P[P_index] = Pvalue;
    printf("Thread (%d, %d) -> P[%d] = %f\n", outRow, outCol, P_index, P[P_index]);
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "Error: %s %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Host function to perform convolution and compare results
void check_result(const float *h_N, const float *h_F, const float *h_P, int rows_N, int cols_N, int rows_f, int cols_f, int r) {
    bool success = true;
    for (int outRow = 0; outRow < rows_N; ++outRow) {
        for (int outCol = 0; outCol < cols_N; ++outCol) {
            float Pvalue = 0.0f;
            for (int fRow = 0; fRow < rows_f; ++fRow) {
                for (int fCol = 0; fCol < cols_f; ++fCol) {
                    int inRow = outRow - r + fRow;
                    int inCol = outCol - r + fCol;
                    if (inRow >= 0 && inRow < rows_N && inCol >= 0 && inCol < cols_N) {
                        Pvalue += h_N[inRow * cols_N + inCol] * h_F[fRow * cols_f + fCol];
                    }
                }
            }
            if (fabs(h_P[outRow * cols_N + outCol] - Pvalue) > 1e-5) {
                success = false;
                printf("Mismatch at (%d, %d): GPU = %f, CPU = %f\n", outRow, outCol, h_P[outRow * cols_N + outCol], Pvalue);
            }
        }
    }
    if (success) {
        printf("Results match!\n");
    } else {
        printf("Results do not match!\n");
    }
}

int main(void) {
    const int rows_N = 16;
    const int cols_N = 16;
    const int rows_f = 3;
    const int cols_f = 3;
    const int r = 1;
    float h_F[rows_f][cols_f] = {
        {1, 1, 1},
        {1, 1, 1},
        {1, 1, 1}
    };

    float h_N[rows_N][cols_N];
    std::srand(std::time(0));
    for (int i = 0; i < rows_N; ++i) {
        for (int j = 0; j < cols_N; ++j) {
            h_N[i][j] = std::rand() % 5 + 1;
        }
    }

    int size_N = rows_N * cols_N * sizeof(float);
    float *d_N, *d_P;
    checkCudaError(hipMalloc((void**)&d_N, size_N), "Failed to allocate device memory for d_N");
    checkCudaError(hipMalloc((void**)&d_P, size_N), "Failed to allocate device memory for d_P");
    checkCudaError(hipMemcpy(d_N, h_N, size_N, hipMemcpyHostToDevice), "Failed to copy data from host to device for d_N");

    // Copy F to constant memory:
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(F), h_F, rows_f * cols_f * sizeof(float)), "Failed to copy data to constant memory for F");

    dim3 dimGrid(rows_N / 4, cols_N / 4, 1);
    dim3 dimBlock(4, 4, 1);
    convolution_2D_basic_kernel<<<dimGrid, dimBlock>>>(d_N, d_P, r, rows_N, cols_N);
    checkCudaError(hipGetLastError(), "Kernel launch failed");
    checkCudaError(hipDeviceSynchronize(), "Kernel execution failed");

    float h_P[rows_N * cols_N];
    checkCudaError(hipMemcpy(h_P, d_P, size_N, hipMemcpyDeviceToHost), "Failed to copy data from device to host for h_P");

    check_result((float*)h_N, (float*)h_F, h_P, rows_N, cols_N, rows_f, cols_f, r);

    checkCudaError(hipFree(d_N), "Failed to free device memory for d_N");
    checkCudaError(hipFree(d_P), "Failed to free device memory for d_P");

    return 0;
}
